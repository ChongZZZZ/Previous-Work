
#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <float.h>
#include <math.h>


__device__ double evaluateBoard(int* board, int size, int AI_role) {
    //inner ones get the most score
    //angles also get the score
    //printf("evaluating\n");
    double score=0;

    double center = (double) ((size-1)/2);

    double half_diagonal= sqrt(2*((double)(size/2)* (double)(size/2)));

    for(int i=0; i< size*size; i++){ //checking current status

        if(board[i]== AI_role){
            // weighted more for corner
            if(i==0 || i==(size-1)|| i == (size*size-1) || i == (size*(size-1))){ 
                score += half_diagonal*0.7;
            } else{ // weighted according to the distance from the center
            int x_int = i / size;
            double x = x_int;
            double y = i % size;
            score = score + half_diagonal - sqrt((x-center)*(x-center)+(y-center)*(y-center));
            }
        }    
    }

    // need to also consider the continuous



    return score;
}

// not full and no winner return 0
// winner == cross return 1
// winner == circle return 2
// full return 3
__device__ int winner_or_full(int* board, int size) {
    
    // Check horizontal and vertical stuff
    for (int i=0; i< size; i++){
        int potential_winner = board[i*size];
        
        bool horizontal= true;
        bool vertical=true;
        // Check for the winner horizontally
        for (int j=0; j < size; j++){
            if (board[i*size+j] != potential_winner){
                horizontal= false;
                break;
            }
        }

        if (horizontal){
            return potential_winner;
        }
        
        // Changing the potential winner to the vertical one
        potential_winner = board[i];
        
        // Check winner in the vertical
        for (int j=0; j < size; j++){
            if (board[j*size+i]!=potential_winner){
                vertical=false;
                break;
            }
        }

        if (vertical){
            return potential_winner;
        }
    }

    // Check // copy the board
      int* copy_board = (int*)malloc(sizeof(int) * size * size);

    for (int n = 0; n < size * size; n++) {
        copy_board[n] = board[n];
    }

    int potential_winner = board[0];
    if (potential_winner != 0){
        bool diagonal = true;

        for (int i=0; i < size; i++){
            if (board[i * size + i] != potential_winner) {
                diagonal = false;
                break;
            }
        }

        if (diagonal) {
            return potential_winner;
        }  
    }

    // Check the other diagonal
    potential_winner = board[size - 1];
    if (potential_winner != 0) {
        bool diagonal = true;

        for (int i = 0; i < size; i++) {
            if (board[i * size + (size - 1 - i)] != potential_winner) {
                diagonal = false;
                break;
            }
        }

        if (diagonal) {
            return potential_winner;
        }
    }

    // Check if the board is filled (cat's scratch)
    for (int i=0; i< size*size; i++){
        if(board[i]==0){
            //no winner and not filled
            return 0;
        }
    }

    return 3;
}

__device__ double MinMax(int* board, int Move, int size, int AI_role, bool Max) {

  int status = winner_or_full(board, size);



  if (status == 1 || status == 2) {  // cross is the winner
    // check whether it is AI
    if (AI_role == status) {
      return 10000;
    } else {
      return -10000;
    }

  } else if (status == 3) {  // board is filled and no winner (a tie)
    return 0;

  } else {
   
    double score = evaluateBoard(board, size, AI_role);
    //printf("evaluating score is %lf\n", score);
    return score;

  }


  // if(blockIdx.x==0 && (threadIdx.x * size + threadIdx.y==1)){
  // printf("xixi: for block %d and thread %d, status: %d\n", blockIdx.x, threadIdx.x * size + threadIdx.y, status);
  // }
 
  if (Max) {
    double bestValmax = -10000;  // evaluating the remaining cells

    // get all the empty cells
    for (int i = 0; i < size * size; i++) {
      if (board[i] == 1 || board[i] == 2) {  //< the board is occupied
        continue;
      }
      // copy the board
      int* copy_board = (int*)malloc(sizeof(int) * size * size);

      for (int n = 0; n < size * size; n++) {
        copy_board[n] = board[n];
      }

      // play the move (AI making the move)
      copy_board[i] = AI_role;

      // if(blockIdx.x==0 && (threadIdx.x * size + threadIdx.y==1)){
      //   for (int n = 0; n < size * size; n++){
      //     printf("In MAx value: %d\n", copy_board[n]);
      //   }
      // }

      bestValmax = fmax(bestValmax, MinMax(copy_board, i, size, AI_role, false));  // calculate the maximum of all the boards.
    }

    // printf("Best Value Max: %lf\n", bestValmax);

    return bestValmax;

  } else {
    double bestValmin = 10000;

    for (int i = 0; i < size * size; i++) {
      if (board[i] == 1 || board[i] == 2) {  // the board is occupied
        continue;
      }

      // copy the board
      int* copy_board = (int*)malloc(sizeof(int) * size * size);

      for (int n = 0; n < size * size; n++) {
        copy_board[n] = board[n];
      }

      // play the move (Human make the move)
      if (AI_role == 1) {
        copy_board[i] = 0;
      } else {
        copy_board[i] = 1;
      }

      // if(blockIdx.x==0 && (threadIdx.x * size + threadIdx.y==1)){
      //   for (int n = 0; n < size * size; n++){
      //     printf("In MIN value: %d\n", copy_board[n]);
      //   }
      // }

      bestValmin = fmin(bestValmin, MinMax(copy_board, i, size, AI_role, true));  // calculate the minimum of all the boards.
    }

    // printf("Best Value Min: %lf\n", bestValmin);
    return bestValmin;
  }
}




__global__ void minimaxKernel(int* board, int  board_size, int human_role, double* result) {
 
  // Make shared variables for calculating the best score
  __shared__ double*  score;
  int num = board_size*board_size;
  __shared__ int* step;

  if (threadIdx.x == 0){
    score = new double[num];
    step = new int[num];
  }

 

  // Each thread is one cell in the potential board
  int current_value = board[blockIdx.x*board_size*board_size+threadIdx.x * board_size + threadIdx.y];

  //printf("for block %d and thread %d, current_value 1: %d\n", blockIdx.x, threadIdx.x * board_size + threadIdx.y, current_value);

  // NOTE: Code doesn't go past here. 

  //The move that I want to play next
  int last_step = threadIdx.x * board_size + threadIdx.y;

  double current_score =  0;

  //If the space is not occupied
  if (current_value == 0) {

    // Copy the board
    int* copy_board = (int*)malloc(sizeof(int) * board_size * board_size);
    // Play another step in each copy board
    for (int n = 0; n < board_size * board_size; n++) {
      copy_board[n] = board[blockIdx.x*board_size*board_size+n];
    }

    // For the current possible move, assign it to the Human so that it
    // can evaluate whether it will be a good move or not
    copy_board[last_step] = human_role;
    //printf("xixi: for block %d and thread %d, current score: %lf and last_step: %d\n", blockIdx.x, threadIdx.x * board_size + threadIdx.y, current_score, last_step);
    if(blockIdx.x==0 && (threadIdx.x * board_size + threadIdx.y==1)){
      for (int n = 0; n < board_size * board_size; n++) {
        //printf("The board revised move: %d\n", copy_board[n]);
      }
    }

    // Default setting
    int AI_role = 0;

    //Get the AI's role
    if (human_role == 1) {
      AI_role = 2;
    } else {
      AI_role = 1;
    }
    //printf("hehe: for block %d and thread %d, current score: %lf and last_step: %d\n", blockIdx.x, threadIdx.x * board_size + threadIdx.y, current_score, last_step);

    // Do the min max algorithm
    current_score = MinMax(copy_board, last_step, board_size, AI_role, true);
    //printf("haha: for block %d and thread %d, current score: %lf and last_step: %d\n", blockIdx.x, threadIdx.x * board_size + threadIdx.y, current_score, last_step);

  } else {
    current_score = 0;
    last_step = -1;
  }

    //printf("for block %d and thread %d, current score: %lf and last_step: %d\n", blockIdx.x, threadIdx.x * board_size + threadIdx.y, current_score, last_step);
    // Sync all of the threads
    __syncthreads();
    score[threadIdx.x * board_size + threadIdx.y]= current_score;
    step[threadIdx.x * board_size + threadIdx.y]= last_step;
    __syncthreads();

    double max_score = -10000;
    if(threadIdx.x == 0 && threadIdx.y==0){
      for(int i = 0 ; i < board_size * board_size; i++){
        
        if(step[i]!=-1){
          if(score[i] > max_score){
            max_score = score[i];
          }
        }
      }
      result[blockIdx.x]= max_score;
    }

    __syncthreads();

    // if(threadIdx.x == 0 && threadIdx.y==0){
    //   printf("current score for the %d block is %lf\n", blockIdx.x, result[blockIdx.x]);
    // }

}

extern "C" __host__ int AI_run(int* board, int board_size, int AI_role){

    // Make array of <x> copies of the board to explore every first possible
    // move that the AI could make against the player
    // storing them all in one array
    int * next_move_board = (int*)malloc (sizeof(int)*board_size*board_size*board_size*board_size);

    

    // moves
    int move[board_size*board_size];

    for (int i=0; i< board_size*board_size; i++){
      move[i]=-1;
    }

    // Counter to count how many spaces the AI could make a move
    int possible_move=0;

    // Put the board copies in the array
    for(int i=0; i < board_size*board_size; i++){
        // If there is an open space
        if(board[i]==0){
            
            for (int n = 0; n < board_size * board_size; n++) {
                next_move_board[possible_move*board_size * board_size+n] = board[n];
            }

            // For the current possible move, assign it to the AI so that it
            // can evaluate whether it will be a good move or not
            next_move_board[possible_move* board_size * board_size+i] =AI_role;

            move[possible_move]= i;

            // Increment counter variable
            possible_move++;
        }


    }

    //printf("Potential move: %d\n", possible_move);

    // for (int i=0; i< board_size*board_size; i++){
    //   printf("CPU: %d \n", next_move_board[board_size*board_size*(possible_move-1)+i]);
    // }

    int human_role = 0;

    if(AI_role==1){
        human_role = 2;
    } else{
        human_role = 1;
    }

    int*  gpu_board;
    double *  gpu_result;
    int* gpu_role;
    int* gpu_board_size;
    

    // Allocate memory on the GPU for the flattened board
    // Allocate space for boards on the GPU
    if (hipMallocManaged(&gpu_board, board_size * board_size * sizeof(int)* possible_move) != hipSuccess) {
        fprintf(stderr, "Failed to allocate boards on GPU\n");
        exit(2);
    }

    // !!!!Score for each step
    if (hipMallocManaged(&gpu_result, possible_move*sizeof(double)) != hipSuccess) {
        fprintf(stderr, "Failed to allocate result on GPU\n");
        exit(2);
    }

    // Allocate role for roleprintf("hahaha\n");
    if (hipMallocManaged(&gpu_role, sizeof(int)) != hipSuccess) {
        fprintf(stderr, "Failed to allocate roles on GPU\n");
        exit(2);
    }

    // Allocate role for board_size
    if (hipMallocManaged(&gpu_board_size, sizeof(int)) != hipSuccess) {
        fprintf(stderr, "Failed to allocate board_size on GPU\n");
        exit(2);
    }


    // Copy the cpu's boards to the gpu with cudaMemcpy
    if (hipMemcpy(gpu_board, next_move_board, board_size * board_size * sizeof(int)* possible_move, hipMemcpyHostToDevice) !=
        hipSuccess) {
        fprintf(stderr, "Failed to copy boards to the GPU\n");
    }

    // Copy the role
    if (hipMemcpy(gpu_role, &human_role, sizeof(int), hipMemcpyHostToDevice) !=
        hipSuccess) {
        fprintf(stderr, "Failed to copy role to the GPU\n");
    }

    // Copy the board size
    if (hipMemcpy(gpu_board_size, &board_size, sizeof(int), hipMemcpyHostToDevice) !=
        hipSuccess) {
        fprintf(stderr, "Failed to copy board_size to the GPU\n");
    }
    
    //Set up block
    dim3 threadsPerBlock(board_size, board_size);

    //dim3 threadsPerBlock(1,3);


    size_t numBlocks = possible_move;

    //size_t numBlocks = 1;

    minimaxKernel<<<numBlocks, threadsPerBlock>>> (gpu_board, *gpu_board_size, *gpu_role, gpu_result);

    
    // Wait for the GPU to finish
    if (hipDeviceSynchronize() != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(hipPeekAtLastError()));
    }


    // place to store the result
    double* cpu_result = (double*) malloc (sizeof(double)*possible_move);

    

    // Transfer the result
    // Transfer the result back to the host
    // Should this be a board ?!
    if (hipMemcpy(cpu_result, gpu_result, possible_move*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "Failed to copy result from the GPU\n");
    }

    // printf("finish from the GPU\n");

    //data store in move suddenly changed ? Maybe some overlap?????


    double best_score = -10000;
    int best_move = 0;
    
  //  for (int i=0; i< possible_move; i++){
  //     printf("%lf\n", cpu_result[i]);
  //   }

    // Find the best move
    for (int i = 0; i < possible_move; i++) {
        if (cpu_result[i] > best_score) {
            best_score = cpu_result[i];
            best_move = move[i];
        }
    }

    // printf("best move: %d for score %lf\n", best_move, best_score);


    // Clean up
    hipFree(gpu_board);
    hipFree(gpu_result);
    hipFree(gpu_role);
    hipFree(gpu_board_size);


    return best_move; 
    
}


